//#include <iostream>

/*

matrix add		            = c = a+b
matrix inner product        = c = a.*b
matrix sub                  = c = a-b
matrix sum                  = s = sum(sum(a)); // reduction shared memory 
matrix multiplication       = c = a*b
matrix inner division       = c = a./b; 
matrix element wise square  = c = a.^2; 
matrix element wise pow     = c = a.^m; 
matrix element wise prod    = c  = a.*m; 
matrix compare and binarise = c = a > const
matrix compare and binarise = c = a < const

*/
// compile and generate ptx file
//nvcc -ptx matOperation.cu --gpu-architecture=compute_61 --gpu-code=sm_61 
        // direk matlan command window dan compile yapabiliriz.
//system('nvcc -ptx matOperation.cu --gpu-architecture=compute_61 --gpu-code=sm_61');
//template <typename T>
        // c = a + b;

#include <hip/hip_runtime.h>
#include <math.h>
                // c = a>const G = grater
__global__ void cudaMatrixCompareAndBinarise_G(double *MatA, const double C,const int nx, const int ny)
{   
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    if (ix < nx && iy < ny)
    {
       if (MatA[idx] > C)
            MatA[idx] = 1;
        else
            MatA[idx] = 0;
    }
}
 // c = a < const L = lower
__global__ void cudaMatrixCompareAndBinarise_L(double *MatA, const double C,const int nx, const int ny)
{   
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    if (ix < nx && iy < ny)
    {
       if (MatA[idx] < C)
            MatA[idx] = 1;
        else
            MatA[idx] = 0;
    }
}
__global__ void sumMatrixGPU(const double *MatA, const double *MatB, double *MatC, const int nx, const int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny)
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }

}

// c  = a.*b; 
__global__ void pointProductMatrixGPU(const double *MatA, const double *MatB, double *MatC, const int nx, const int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny)
    {
        MatC[idx] = MatA[idx] * MatB[idx];
    }
}
// c  = a.*m; 
__global__ void matrixProductConstGPU(const double *MatA, const double constB, double *MatB, const int nx, const int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny)
    {
        MatB[idx] = MatA[idx] * constB;
    }
}
// c  = a.^m; 
__global__ void matrixCalculatePowGPU(const double *MatA, const double constB, double *MatB, const int nx, const int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny)
    {
        MatB[idx] = pow(MatA[idx],constB);
    }
}
// c  = n*(a.^m);  

__global__ void matrixCalculatePowAndMultGPU(const double *MatA, const double constB,const double constC, double *MatB, const int nx, const int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny)
    {
        MatB[idx] = pow(MatA[idx],constB);
        MatB[idx] = MatB[idx] * constC;
    }
}
// c = a - b;
__global__ void subtractMatrixGPU(const double *MatA, const double *MatB, double *MatC, const int nx, const int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny)
    {
        MatC[idx] = MatA[idx] - MatB[idx];
    }
}
// s = sum(sum(a));
__global__ void reduceSum( double* g_odata,  const double* g_idata,  const double len) {
	extern __shared__  double sdata[];

	// each thread loads one element from global to shared mem
	// Do the first stage of the reduction on the global-to-shared load step
	// This reduces the previous inefficiency of having half of the threads being
	//  inactive on the first for-loop iteration below (previous first step of reduction)
	// Previously, only less than or equal to 512 out of 1024 threads in a block are active.
	// Now, all 512 threads in a block are active from the start
	 unsigned int tid = threadIdx.x;
	 unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	sdata[tid] = 0;

	if (i < len)
	{
		sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
	}

	__syncthreads();

	// do reduction in shared mem
	// this loop now starts with s = 512 / 2 = 256
	for ( unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid < 32)
	{
		sdata[tid] += sdata[tid + 32];
		sdata[tid] += sdata[tid + 16];
		sdata[tid] += sdata[tid + 8];
		sdata[tid] += sdata[tid + 4];
		sdata[tid] += sdata[tid + 2];
		sdata[tid] += sdata[tid + 1];
	}

	// write result for this block to global mem
	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
}

// reduceSumV2
__global__ void reduceSmemV2(const double *g_idata, double *g_odata, unsigned int n)
{
    // static shared memory
    __shared__ double smem[128];

    // set thread ID
    unsigned int tid = threadIdx.x;

    // global index, 4 blocks of input data processed at a time
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // unrolling 4 blocks
    double tmpSum = 0;

    // boundary check
    if (idx + 4 * blockDim.x <= n)
    {
        double a1 = g_idata[idx];
        double a2 = g_idata[idx + blockDim.x];
        double a3 = g_idata[idx + 2 * blockDim.x];
        double a4 = g_idata[idx + 3 * blockDim.x];
        tmpSum = a1 + a2 + a3 + a4;
    }

    smem[tid] = tmpSum;
    __syncthreads();

    // in-place reduction in shared memory
    if (blockDim.x >= 1024 && tid < 512) smem[tid] += smem[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256)  smem[tid] += smem[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128)  smem[tid] += smem[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64)   smem[tid] += smem[tid + 64];

    __syncthreads();

    // unrolling warp
    if (tid < 32)
    {
        volatile double *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}
// c = a*b matrix multiplication 

__global__ void MatrixMulKernel(double* Md, double* Nd, double* Pd, int Width)
{
     __shared__ double Mds[32][32];
     __shared__ double Nds[32][32];
     int bx = blockIdx.x; int by = blockIdx.y;
     int tx = threadIdx.x; int ty = threadIdx.y;
    // Identify the row and column of the Pd element to work on
     int Row = by * 32 + ty;
     int Col = bx * 32 + tx;
     double Pvalue = 0;
    // Loop over the Md and Nd tiles required to compute the Pd element
     for (int m = 0; m < Width/32; ++m) {
    // Collaborative loading of Md and Nd tiles into shared memory
     Mds[ty][tx] = Md[Row*Width + (m*32 + tx)];
     Nds[ty][tx] = Nd[Col + (m*32 + ty)*Width];
     __syncthreads();
     for (int k = 0; k < 32; ++k)
       Pvalue += Mds[ty][k] * Nds[k][tx];
     __syncthreads();
     }
     Pd[Row*Width+Col] = Pvalue;
}

// c  = a./b; 
__global__ void pointDivisionMatrixGPU(const double *MatA, const double *MatB, double *MatC, const int nx, const int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny)
    {
        MatC[idx] = MatA[idx] / MatB[idx];
    }
}
// c  = a.^2  
__global__ void pointSquareMatrixGPU(const double *MatA, double *MatC, const int nx, const int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny)
    {
        MatC[idx] = MatA[idx] * MatA[idx];
    }
}