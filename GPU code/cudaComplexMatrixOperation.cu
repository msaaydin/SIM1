//#include <cuComplex.h>
 
//system('nvcc -ptx cudaComplexMatrixOperation.cu --gpu-architecture=compute_61 --gpu-code=sm_61');
/* Double precision */
typedef double2 hipDoubleComplex;

#include <hip/hip_runtime.h>
#include <math.h>
__device__  double hipCreal (hipDoubleComplex x) 
{ 
    return x.x; 
}
__device__ double hipCimag (hipDoubleComplex x) 
{ 
    return x.y; 
}

__device__ hipDoubleComplex make_hipDoubleComplex(double r, double i)
{
    hipDoubleComplex res;
    res.x = r;
    res.y = i;
    return res;
}
// c = a+b a ve b birer complex number
__device__ hipDoubleComplex hipCadd(hipDoubleComplex x, hipDoubleComplex y)
{
    return make_hipDoubleComplex (hipCreal(x) + hipCreal(y), 
                                 hipCimag(x) + hipCimag(y));
}
// c = conj(a) a bir complex number
__device__ hipDoubleComplex hipConj(hipDoubleComplex x)
{
    return make_hipDoubleComplex (hipCreal(x), -hipCimag(x));
}
// c = a-b a ve b birer complex number
__device__ hipDoubleComplex hipCsub(hipDoubleComplex x, hipDoubleComplex y)
{
    return make_hipDoubleComplex (hipCreal(x) - hipCreal(y), 
                                 hipCimag(x) - hipCimag(y));
}

/* This implementation could suffer from intermediate overflow even though
 * the final result would be in range. However, various implementations do
 * not guard against this (presumably to avoid losing performance), so we 
 * don't do it either to stay competitive.
 */
 // c = a*b a ve b birer complex number
__device__ hipDoubleComplex hipCmul(hipDoubleComplex x,hipDoubleComplex y)
{
    hipDoubleComplex prod;
    prod = make_hipDoubleComplex ((hipCreal(x) * hipCreal(y)) - 
                                 (hipCimag(x) * hipCimag(y)),
                                 (hipCreal(x) * hipCimag(y)) + 
                                 (hipCimag(x) * hipCreal(y)));
    return prod;
}

/* This implementation guards against intermediate underflow and overflow
 * by scaling. Such guarded implementations are usually the default for
 * complex library implementations, with some also offering an unguarded,
 * faster version.
 */
 // c = a/b a ve b birer complex number
__device__ hipDoubleComplex hipCdiv(hipDoubleComplex x, hipDoubleComplex y)
{
    hipDoubleComplex quot;
    double s = (fabs(hipCreal(y))) + (fabs(hipCimag(y)));
    double oos = 1.0 / s;
    double ars = hipCreal(x) * oos;
    double ais = hipCimag(x) * oos;
    double brs = hipCreal(y) * oos;
    double bis = hipCimag(y) * oos;
    s = (brs * brs) + (bis * bis);
    oos = 1.0 / s;
    quot = make_hipDoubleComplex (((ars * brs) + (ais * bis)) * oos,
                                 ((ais * brs) - (ars * bis)) * oos);
    return quot;
}

/* This implementation guards against intermediate underflow and overflow
 * by scaling. Otherwise we would lose half the exponent range. There are
 * various ways of doing guarded computation. For now chose the simplest
 * and fastest solution, however this may suffer from inaccuracies if sqrt
 * and division are not IEEE compliant.
 */
 // c = abs(a) a bir complex number
__device__ double hipCabs (hipDoubleComplex x)
{
    double a = hipCreal(x);
    double b = hipCimag(x);
    double v, w, t;
    a = fabs(a);
    b = fabs(b);
    if (a > b) {
        v = a;
        w = b; 
    } else {
        v = b;
        w = a;
    }
    t = w / v;
    t = 1.0 + t * t;
    t = v * sqrt(t);
    if ((v == 0.0) || 
        (v > 1.79769313486231570e+308) || (w > 1.79769313486231570e+308)) {
        t = v + w;
    }
    return t;
}
// global kernel function call all math operations..
// c = a/b
__global__ void cudaComplexDiv(const double2 *MatA, const double2 *MatB, double2 *MatC,const int nx, const int ny)
{   
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    if (ix < nx && iy < ny)
    {
        MatC[idx] = hipCdiv(MatA[idx], MatB[idx]);
    }
	//*MatC = cuCdiv(MatA[0],MatB[0]);    
} 

// c = a+b
__global__ void cudaComplexAdd(const double2 *MatA, const double2 *MatB, double2 *MatC,const int nx, const int ny)
{   
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    if (ix < nx && iy < ny)
    {
        MatC[idx] = hipCadd(MatA[idx], MatB[idx]);
    }
	//*MatC = cuCdiv(MatA[0],MatB[0]);    
}

// c = a-b
__global__ void cudaComplexSub(const double2 *MatA, const double2 *MatB, double2 *MatC,const int nx, const int ny)
{   
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    if (ix < nx && iy < ny)
    {
        MatC[idx] = hipCsub(MatA[idx], MatB[idx]);
    }
	//*MatC = cuCdiv(MatA[0],MatB[0]);    
}
      
// c = a*b
__global__ void cudaComplexMul(const double2 *MatA, const double2 *MatB, double2 *MatC,const int nx, const int ny)
{   
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    if (ix < nx && iy < ny)
    {
        MatC[idx] = hipCmul(MatA[idx], MatB[idx]);
    }
	//*MatC = cuCdiv(MatA[0],MatB[0]);    
}
// c = conj(a)
__global__ void cudaComplexConj(double2 *MatA,const int nx, const int ny)
{   
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    if (ix < nx && iy < ny)
    {
        MatA[idx] = hipConj(MatA[idx]);
    }
	//*MatC = cuCdiv(MatA[0],MatB[0]);    
}
// c = abs(a)
__global__ void cudaComplexAbs(const double2 *MatA, double *MatC,const int nx, const int ny)
{   
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    if (ix < nx && iy < ny)
    {
        MatC[idx] = hipCabs(MatA[idx]);
    }
	//*MatC = cuCdiv(MatA[0],MatB[0]);    
}

// c = a>const
__global__ void cudaMakeBinaryMatrix(const double *MatA, double *MatC, const double C,const int nx, const int ny)
{   
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    if (ix < nx && iy < ny)
    {
       if (MatA[idx] > C)
            MatC[idx] = 1;
        else
            MatC[idx] = 0;
    }
	//*MatC = cuCdiv(MatA[0],MatB[0]);    
}
// c = a > const
__global__ void cudaMakeBinaryMatrix2(double *MatA, const double C,const int nx, const int ny)
{   
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    if (ix < nx && iy < ny)
    {
        if (MatA[idx] > C)
            MatA[idx] = 1;
        else
            MatA[idx] = 0;
            
    }
	//*MatC = cuCdiv(MatA[0],MatB[0]);    
}